#include "hip/hip_runtime.h"
#include<iostream>
#include<math.h>
#include<typeinfo>
#include<boost/detail/sp_typeinfo.hpp>
#include "CudaLE.h"

using namespace CudaLE;

#define CURRENT_FUNCTION __PRETTY_FUNCTION__

// const Coordinate Type type = COORD_CYLINDRICAL;
// struct ScaleFuncBase
// {
//     HOST_DEVICE virtual double operator() (double x1, double x2 = 0.0, double x3 = 0.0, double x4 = 0.0) = 0;
//     HOST_DEVICE virtual ~ScaleFuncBase() {}
// };

// // typedef UnaryOp<Id, double> Var;
// template <typename Func>
// struct ScaleFunc : public ScaleFuncBase
// {
//     Func h;
//     HOST_DEVICE ScaleFunc (Func func) : h(func) {}
//     HOST_DEVICE virtual double operator() (double x1, double x2 = 0.0, double x3 = 0.0, double x4 = 0.0) {
//         return h(x1, x2);
//     }
//     HOST_DEVICE virtual ~ScaleFunc() {}
// }; // ----- end of class Scales -----

// template <typename F>
// struct Finvoke
// {
//     HOST_DEVICE static double invoke (F func, double x1, double x2, double x3, double x4 = 0.0) {
//         F f(func);
//         return f(x1, x2, x3);
//     }
// };

struct FunctionBase
{
    HOST_DEVICE virtual double operator() (double x1, double x2 = 0.0, double x3 = 0.0, double x4 = 0.0) = 0;
    HOST_DEVICE virtual ~FunctionBase() {}
    // template<typename F>
    // F getFunc(FunctionBase*);
};

template <typename F>
struct Function : public FunctionBase
{
    F f;
    HOST_DEVICE Function(F func) : f(func) {}
    HOST_DEVICE virtual double operator() (double x1, double x2 = 0.0, double x3 = 0.0, double x4 = 0.0) {
        return f(x1, x2, x3);
    }
    HOST_DEVICE virtual ~Function() {}
};

// template <typename F>
// struct func_wrapper : public F
// {
//     func_wrapper(F f) : F(f) {}
// };

// struct function_buffer
// {
//     void * f;

//     struct type_t {
//         // const boost::detail::sp_typeinfo* type;
//         std::type_info* type;
//     } type;
// };

// template<typename Functor>
// struct functor_manager
// {
//     HOST_DEVICE static inline void
//     manage(const function_buffer& in_buffer) {
//         Functor* f = static_cast<Functor*>(in_buffer.f);
//         delete f;
//         // in_buffer.f = 0;
//     }
// };

// template <typename F>
// struct Invoker
// {
//     HOST_DEVICE static double invoke(function_buffer& functor, double x1, double x2, double x3, double x4 = 0.0) {
//         F* f;
//         f = reinterpret_cast<F*>(functor.f);
//         return (*f)(x1, x2, x3);
//     }
// };

// struct Get_Invoker
// {
//     template<typename F>
//     struct apply
//     {
//         typedef Invoker<F> invoker_type;
//         typedef functor_manager<F> manage_type;
//     };
// };

// struct vtable_base
// {
//     void (*manager)(const function_buffer& in_buffer);
// };

// struct vtable
// {
//     typedef double (*invoker_type)(function_buffer&,
//                                    double, double, double);

//     void (*manager)(const function_buffer& in_buffer);
    
//     template<typename F>
//     HOST_DEVICE void
//     assign_functor(F f, function_buffer& functor) const {
//         functor.f = new F(f);
//     }

//     void
//     clean (function_buffer& functor) const {
//     }

//     invoker_type invoker;
// };

// struct FunctionTest
// {
//     function_buffer functor;
//     vtable* vtable_;

//     template <typename F>
//     HOST_DEVICE FunctionTest(F func) {
//         assign(func);
//     }

//     HOST_DEVICE double operator() (double x1, double x2 = 0.0, double x3 = 0.0, double x4 = 0.0) {
//         return vtable_ -> invoker(functor, x1, x2, x3);
//     }

//     template <typename F>
//     HOST_DEVICE void assign(F f) {
//         typedef typename Get_Invoker::template apply<F> handler_type;
//         typedef typename handler_type::invoker_type invoker_type;
//         typedef typename handler_type::manage_type manager_type;

//         // static const vtable stored_vtable = { &invoker_type::invoke, &manager_type::manage };
//         vtable stored_vtable;
//         stored_vtable.invoker = &invoker_type::invoke;
//         stored_vtable.manager = &manager_type::manage;
//         stored_vtable.assign_functor(f, functor);
//         // functor.type.type = &typeid(F);
//         size_t value = reinterpret_cast<size_t>(&stored_vtable);
//         vtable_ = reinterpret_cast<vtable*>(value);
//     }
// };

// template <typename T>
// ScaleFuncBase* MakeScaleFunc(T func) {
//     return (new ScaleFunc<T>(func));
// }

template <typename T>
FunctionBase* MakeFunction(T func) {
    FunctionBase* newFunc = new Function<T>(func);
    // newFunc -> invoker_type = &Finvoke<T>::invoke;
    return newFunc;
}

// template<class T> struct sp_typeid_ {
//     static char const * name() {
//         return CURRENT_FUNCTION;
//     }
// };

// typedef UnaryOp<Id, double> Var;
template <typename F>
__global__ void test (F func, double x) {
    // printf("%s\n", typeid(int).name());
    // printf("%s\n", sp_typeid_::name());
    printf("%e\n", func(x));
}

int main () 
{
    // FunctionBase* myF = MakeFunction(_sin(_1) + _2);
    FunctionBase& myG = *MakeFunction(_sin(_2) * _exp(_1) + _2 + _cos(_3));
    // typeof(*myG) mytest;
    // Function func(_sin(_1));
    // std::cout << "Testing vtable" << func(2.0) << std::endl;
    // std::cout << typeid(_sin(_1)).name() << std::endl;
    // Var _x(1), _y(2);
    // (_sin(_1) + _2)::op_type myF = MakeFunc(_sin(_1) + _2);
    // auto myF = _sin(_1) + _2;
    // std::cout << sin(2.0) + 3.0 << std::endl;
    // std::cout << (_log(_1) * _exp(_2) + _3)(2.0, -1, 1.0) << std::endl;
    // std::cout << _sin(_pow<2>(_1) + _2)(2.0, 3.0) << std::endl;
    // std::cout << (myG -> getFunc(myG))(2.0, 3.0) << std::endl;
    std::cout << myG(1.0, 2.0, 3.0) << std::endl;
    delete &myG;
    // std::cout << (myF * _sin(_2))(1.0, 2.0) << std::endl;
    // std::cout << (_exp(myG))(1.0, 2.0) << std::endl;
    // std::cout << sizeof(myG) << std::endl;
    // test<<<1, 1>>>(_sin(_1), 2.0);
    // test<<<1, 1>>>((* dynamic_cast<Function<typeof(_sin(_2) + _1)>*>(myG)).f, 2.0);
    hipDeviceSynchronize();
    // delete myF;
    // delete myG;
    return 0;
}
