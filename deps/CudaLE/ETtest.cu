

#include <hip/hip_runtime.h>
#include<math.h>
    
#ifdef __HIPCC__
#define HOST_DEVICE __host__ __device__
#else
#define HOST_DEVICE
#endif
    
struct Id {};
    
template <typename Op, typename Left, typename Right>
struct BinaryOp
{
    Left left;
    Right right;
    HOST_DEVICE BinaryOp(Left t1, Right t2) : left(t1), right(t2) {}
    
    HOST_DEVICE double operator() (double x1, double x2 = 0.0, double x3 = 0.0, double x4 = 0.0) {
        return Op::apply(left(x1, x2, x3), right(x1, x2, x3));
    }
};
    
template <typename Op, typename Arg>
struct UnaryOp
{
    Arg arg;
    HOST_DEVICE UnaryOp(Arg t1) : arg(t1) {}
    
    HOST_DEVICE double operator() (double x1, double x2 = 0.0, double x3 = 0.0, double x4 = 0.0) {
        return Op::apply(arg(x1, x2, x3));
    }
};
    
template <int argnum>
struct Var
{
    HOST_DEVICE Var() {}
    HOST_DEVICE double operator() (double x1, double x2 = 0.0, double x3 = 0.0, double x4 = 0.0) {
        if (1 == argnum) return x1;
        else if (2 == argnum) return x2;
        else return x3;
    }
};
    
struct Sin
{
    HOST_DEVICE static double apply(double x) {
        return sin(x);
    }
};
    
struct Plus
{
    HOST_DEVICE static double apply(double a, double b) {
        return a + b;
    }
};
    
template <typename Left, typename Right>
BinaryOp<Plus, Left, Right> operator+ (Left lhs, Right rhs) {
    return BinaryOp<Plus, Left, Right>(lhs, rhs);
}
    
template <typename Arg>
UnaryOp<Sin, Arg> _sin(Arg arg) {
    return UnaryOp<Sin, Arg>(arg);
}
    
template <class T>
__global__ void test(T func, double x, double y, double z = 0.0) {
    printf("%e\n", func(x, y));
}

Var<1> _x;
Var<2> _y;
    
int main () 
{
    test<<<1, 1>>>(_sin(_x) + _y, 1.0, 2.0);
    hipDeviceSynchronize();  // Needed or the host will return before kernel is finished
    return 0;
}
